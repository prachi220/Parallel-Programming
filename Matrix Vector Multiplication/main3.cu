#include "hip/hip_runtime.h"
#include <mpi.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>


#define WARP_SIZE 32
#define BLOCK_SIZE 1024


#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}


__global__  void matrixMultiply(int num_rows, int * ptr , int * indices, unsigned long int * data , unsigned long int * x, unsigned long int * y){
	 

//	int  row = blockDim.x * blockIdx.x + threadIdx.x;
//	if(row < num_rows ){
//		unsigned long int dot = 0;
//		int  row_start = ptr[row];
//		int  row_end    = ptr[row +1];
//		for (int jj = row_start; jj < row_end; jj++)
//			dot += data[jj] * x[indices[jj]];
//		y[row] += dot;
//	}

	 	__shared__ unsigned long int sdata[1024];
     __shared__ int ptrs[1024/WARP_SIZE][2];
    
     const int thread_id   = BLOCK_SIZE * blockIdx.x + threadIdx.x;  // global thread index
     const int thread_lane = threadIdx.x & (WARP_SIZE-1);            // thread index within the warp
     const int warp_id     = thread_id   / WARP_SIZE;                // global warp index
     const int warp_lane   = threadIdx.x / WARP_SIZE;                // warp index within the CTA
     const int num_warps   = (BLOCK_SIZE / WARP_SIZE) * gridDim.x;   // total number of active warps

     for(int row = warp_id; row < num_rows; row += num_warps){
         // use two threads to fetch ptr[row] and ptr[row+1]
         // this is considerably faster than the more straightforward option
         if(thread_lane < 2)
             ptrs[warp_lane][thread_lane] = ptr[row + thread_lane];
         const int row_start = ptrs[warp_lane][0]; //same as: row_start = ptr[row];
         const int row_end   = ptrs[warp_lane][1]; //same as: row_end   = ptr[row+1];

         // compute local sum
         sdata[threadIdx.x] = 0;
         for(int jj = row_start + thread_lane; jj < row_end; jj += WARP_SIZE)
             sdata[threadIdx.x] += data[jj] * x[indices[jj]];

         // reduce local sums to row sum (ASSUME: warpsize 32)
         if (thread_lane < 16) { sdata[threadIdx.x] += sdata[threadIdx.x + 16]; __syncthreads(); }
         if (thread_lane <  8) { sdata[threadIdx.x] += sdata[threadIdx.x +  8]; __syncthreads(); }
         if (thread_lane <  4) { sdata[threadIdx.x] += sdata[threadIdx.x +  4]; __syncthreads(); }
         if (thread_lane <  2) { sdata[threadIdx.x] += sdata[threadIdx.x +  2]; __syncthreads(); }
         if (thread_lane <  1) { sdata[threadIdx.x] += sdata[threadIdx.x +  1]; __syncthreads(); }

         // first thread writes warp result
         if (thread_lane == 0)
             y[row] += sdata[threadIdx.x];
     }
}


int main (int argc, char **argv)
{
    int rank, nprocs;
    MPI_Init (&argc, &argv);
    MPI_Comm_rank (MPI_COMM_WORLD, &rank);
    MPI_Comm_size (MPI_COMM_WORLD, &nprocs);

    FILE* file = fopen(argv[1], "r"); /* should check the result */
  	char c;
  	char buffer[256] ;
  	int index_buf = 0;
  	int line_num=0;
  	int take_input=0;
  	int take_input_A=0;
  	int take_input_B=0;
  	int ARRAY_SIZE;
  	int num_rows;
  	unsigned long int *B, *device_B;
  	int cpu_counter=0, num_non_zero=0;
  	unsigned long int *result_host, *result_gpu;
  	//int result;
  	int Acol=0, Arow=0;
  	int num, line_num_B=0;
  	int  totalBlocks;
  	unsigned long int *data, *device_data;
  	int *indices, *rows, *ptr, *device_indices, *device_rows, *device_ptr;

  	while ((c = getc(file)) != EOF)
    {
    	//printf("c= %c\n", c );
    	if(line_num==0){
    		if(c=='\n'){
				line_num++;
			}
    	}
    	if(line_num == 1){
		   	if(c == ' '){
		   		take_input= 1;
		   	}
		   	if(take_input){
		   		if(c == '\n'){
		   			ARRAY_SIZE = atoi(buffer) ;
		   			
		   			num_rows=ARRAY_SIZE/nprocs;
				if(rank==(nprocs-1)){
					num_rows = num_rows+(ARRAY_SIZE%nprocs);
				}
        			memset(buffer, 0, sizeof(buffer));
        			index_buf = 0 ;
        			line_num++;
        			B = (unsigned long int  *)malloc (sizeof(unsigned long int )*ARRAY_SIZE);
        			data = (unsigned long int  *)malloc (sizeof(unsigned long int )*ARRAY_SIZE*num_rows);
        			indices = (int *)malloc (sizeof(int)*ARRAY_SIZE*num_rows);
        			rows = (int *)malloc (sizeof(int)*ARRAY_SIZE*num_rows);
        			ptr = (int *)malloc (sizeof(int)*(num_rows+1));

		   		}
		   		else{
		   			buffer[index_buf] = c ;
		   			index_buf++;
		   		}
		   	}	    	
		}
		else{

			if(line_num > 1){

				if(c == 'A'){
					take_input_A=1;
					take_input=0;
				}
				if(take_input_A){
					if(line_num==2 && (c=='\n')){
						line_num++;
					}else{
					//printf("line_num: %d\n", line_num);
						if(c == ' '){
							
							num = atoi(buffer) ;
							
							if(Acol==0){
								Arow=num;
							}
							// printf("rank: %d", rank);
							// printf(" Arow: %d", Arow);
							// printf(" Acol: %d", Acol);
							// printf(" num: %d\n", num);
							if((Arow >= (ARRAY_SIZE/nprocs)*rank) && (Arow < (((ARRAY_SIZE/nprocs)*rank)+num_rows))){
								if(Acol==0){
									rows[num_non_zero]=Arow-(rank*(ARRAY_SIZE/nprocs));
								}
								if(Acol==1){
									indices[num_non_zero]=num;
								}
							}
							Acol++;
							memset(buffer, 0, sizeof(buffer));
	    					index_buf = 0 ;
						}else{
							if(c== '\n'){
								unsigned long int num1 = strtoul(buffer, NULL, 0) ;
								// printf("rank: %d", rank);
								// printf(" Arow: %d", Arow);
								// printf(" Acol: %d", Acol);
								// printf(" num: %d\n", num);
								if((Arow >= (ARRAY_SIZE/nprocs)*rank) && (Arow < (((ARRAY_SIZE/nprocs)*rank)+num_rows))){
									data[num_non_zero]=num1;
									num_non_zero++;
								}
								Acol=0;
								memset(buffer, 0, sizeof(buffer));
		    					index_buf = 0 ;
		    					line_num++;
							}else{
								if(c != 'A'){
									buffer[index_buf] = c;
									index_buf++;
								}
							}
						}
					}
				}

				if(c == 'B'){
					take_input_B=1;
					take_input_A=0;
					line_num_B=line_num;
					index_buf=0;
				}
				if(take_input_B){
					// printf("c= %c\n", c );
					if(c== '\n'){
						if(line_num==line_num_B){
							line_num++;
						}else{
							unsigned long int num1 = strtoul(buffer, NULL, 0) ;
							B[line_num-line_num_B-1]=num1;
	        				memset(buffer, 0, sizeof(buffer));
	        				index_buf = 0 ;
	       					line_num++;
						}		
					}
					else{
						if(c != 'B'){
								buffer[index_buf] = c;
								index_buf++;
						}	
					}
				}
			}
		}
	}
	unsigned long int num1 = strtoul(buffer, NULL, 0) ;
	//printf(" num: %d\n", num);
	B[line_num-line_num_B-1]=num1;
	memset(buffer, 0, sizeof(buffer));
	index_buf = 0 ;
	fclose(file);

	//printf(" Printing data here: ");
	//printf("rank: %d", rank);
	//for(int i=0;i<num_non_zero;i++){
	//	printf(" %d", data[i] );
	//}
	//printf("\n");

	int count=0;
	int ptr_counter=1;
	ptr[0]=0;
	for(int n=0; n < num_rows; n++){
		for(int i=0;i<num_non_zero;i++){
			if(n==rows[i]){
				count++;
			}
		}
		ptr[ptr_counter]= ptr[ptr_counter-1]+count;
		ptr_counter++;
		count=0;
	}


	result_host = (unsigned long int *)malloc(sizeof(unsigned long int) * num_rows);
    //GPU

	gpuErrChk(hipMalloc((void**)&device_data, sizeof(unsigned long int ) * ARRAY_SIZE * num_rows));
    gpuErrChk(hipMalloc((void**)&device_B, sizeof(unsigned long int ) * ARRAY_SIZE));
    gpuErrChk(hipMalloc((void**)&device_indices, sizeof(int) * ARRAY_SIZE * num_rows));
    gpuErrChk(hipMalloc((void**)&device_ptr, sizeof(int) * (num_rows+1)));
	gpuErrChk(hipMalloc((void**)&result_gpu, sizeof(unsigned long int) * num_rows));
	
	gpuErrChk(hipMemcpy(device_data, data, sizeof(unsigned long int )*ARRAY_SIZE*num_rows, hipMemcpyHostToDevice));
	gpuErrChk(hipMemcpy(device_B, B, sizeof(unsigned long int ) * ARRAY_SIZE, hipMemcpyHostToDevice));
	gpuErrChk(hipMemcpy(device_indices, indices, sizeof(int) * ARRAY_SIZE * num_rows, hipMemcpyHostToDevice));
	gpuErrChk(hipMemcpy(device_ptr, ptr, sizeof(int) * (num_rows+1), hipMemcpyHostToDevice));
	//10,1024
	//int non_zero_rows=ptr_counter-1;

	memset(result_host, 0, sizeof(result_host));
	hipMemset(result_gpu, 0, sizeof(unsigned long int) * num_rows);
	int NUM_BLOCKS = ((ARRAY_SIZE * num_rows)/1024) + 1;
	matrixMultiply<<<NUM_BLOCKS, 1024>>>(num_rows, device_ptr, device_indices,device_data, device_B, result_gpu);

    hipDeviceSynchronize(); 
    

    //@@ Copy the GPU memory back to the CPU here
    gpuErrChk(hipMemcpy(result_host, result_gpu, sizeof(unsigned long int) * num_rows, hipMemcpyDeviceToHost));
     

    //@@ Free the GPU memory here
    //hipFree(device_data);
    hipFree(device_B);
    hipFree(device_indices);
    hipFree(device_ptr);
    hipFree(result_gpu);

   // printf(" gather_count : %d\n",*gather_count);
	//  printf(" num rows : %d\n",num_rows);
	//for(int i=0;i<num_rows;i++){
	//	printf("rank : %d value of result : %ld \n", rank, result_host[i] );
	//}
	//printf("\n"); 

	MPI_Barrier(MPI_COMM_WORLD);
	unsigned long int * final_answer = (unsigned long int *)malloc(sizeof(unsigned long int) * ARRAY_SIZE);
	int *gather_count = (int*)malloc(sizeof(int));
	
	//num_rows=ARRAY_SIZE/nprocs;
	//if(rank==(nprocs-1)){
	//	num_rows = num_rows+ARRAY_SIZE%nprocs;
	//}
	*gather_count=ARRAY_SIZE/nprocs; 
	int last= nprocs-1;

	
	
	MPI_Gather(result_host, *gather_count, MPI_UNSIGNED_LONG, final_answer, *gather_count, MPI_UNSIGNED_LONG, last, MPI_COMM_WORLD);

	if(rank==last){
		int remaining = ARRAY_SIZE%nprocs;
		// printf(" remaining : %d\n",remaining);
		for(int i= remaining; i> 0; i--){
		//printf("rank : %llu value of answer : %ld \n", rank, final_answer[i] );
			final_answer[ARRAY_SIZE-i]=result_host[num_rows-i];
		}
	}

	if(rank==last){
		//printf(" Printing final result here");
		//for(int i=0;i<ARRAY_SIZE;i++){
		//	printf("rank : %llu value of answer : %ld \n", rank, final_answer[i] );
		//}

		FILE *f1 = fopen(argv[2], "wb");
	  	if (f1 == NULL)
	  	{
	      printf("Error opening file!\n");
	      exit(1);
	  	}
	  	for(int i=0 ; i< ARRAY_SIZE ; i++){
	    	fprintf(f1, "%ld\n", final_answer[i]);
	  	}

	  	fclose(f1) ;
  	}

  	free(B);
	free(data);
	free(indices);
	free(rows);
	free(ptr);
	free(result_host);
	free(final_answer);

    MPI_Finalize();
    return 0;
    
}

